#include "hip/hip_runtime.h"
/*
 * Copyright: See LICENSE file that comes with this distribution
 *
 */

#include "cupp/deviceT/memory1d.h"

#include "kernel_t.h"

__global__ void global_function (cupp::deviceT::memory1d<int>& p) {
	p[threadIdx.x]*=2;
}

kernelT get_kernel() {
	return (kernelT)global_function;
}
