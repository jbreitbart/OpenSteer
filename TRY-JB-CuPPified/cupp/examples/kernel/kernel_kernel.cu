#include "hip/hip_runtime.h"
/*
 * Copyright: See LICENSE file that comes with this distribution
 *
 */

#include "kernel_t.h"

__global__ void global_function (const int i, int &j) {
	j = i;
}

kernelT get_kernel() {
	return (kernelT)global_function;
}
