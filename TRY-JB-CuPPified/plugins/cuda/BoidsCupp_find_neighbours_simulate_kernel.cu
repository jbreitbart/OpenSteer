#include "hip/hip_runtime.h"
#include "cupp/deviceT/vector.h"
#include "cupp/common.h"

#include "OpenSteer/deviceT/Vec3.h"
#include "OpenSteer/CuPPConfig.h"
#include "OpenSteer/kernels.h"

using OpenSteer::deviceT::Vec3;


#if 0
// paper - store version
__global__ void find_neighbours_simulate (const cupp::deviceT::vector< Vec3  > &positions_,
                                          const cupp::deviceT::vector< Vec3  > &forwards_,
                                                cupp::deviceT::vector< Vec3 >  &steering_results_)
{
	// make local copies of our references
	__shared__ cupp::deviceT::vector< Vec3 > positions;
	positions = positions_;
	__shared__ cupp::deviceT::vector< Vec3 > forwards;
	forwards = forwards_;
	__shared__ cupp::deviceT::vector< Vec3 > steering_results;
	steering_results = steering_results_;

	// constants that are needed below
	const unsigned int my_index        = blockIdx.x*blockDim.x + threadIdx.x;
	
	__shared__ unsigned int number_of_boids;
	number_of_boids = gridDim.x*blockDim.x; // number of boids == number of threads

	int neighbours_found = 0;
	
	const Vec3 position = positions[my_index];
	const Vec3 forward  = forwards[my_index];
	
	int   neighbours[neighbour_size_max];
	float neighbours_distance_squared[neighbour_size_max];
	Vec3  neighbours_offset[neighbour_size_max];
	Vec3  neighbours_forward[neighbour_size_max];
	Vec3  neighbours_position[neighbour_size_max];
	
	for (int i=0; i<neighbour_size_max; ++i) {
		neighbours[i]=-1;
	}

	for (int base=0; base < number_of_boids; base+=threads_per_block) {
		__shared__ Vec3 s_positions[threads_per_block];
		s_positions[threadIdx.x] = positions[base + threadIdx.x];
		__syncthreads();

		int i=0;
		while (i < threads_per_block) {
			const Vec3 offset = position - s_positions[i];
			const float d2 = offset.lengthSquared();
			const int cur_index = base + i;
			
			if (d2 < r2 && cur_index != my_index) {
				if (neighbours_found < neighbour_size_max) {
					neighbours[neighbours_found]          = cur_index;
					neighbours_distance_squared[neighbours_found] = d2;
					neighbours_offset[neighbours_found]   = offset;
					neighbours_position[neighbours_found] = s_positions[i];
					++neighbours_found;
				} else {
					float max_neighbour_distance = 0.0f;
					int max_neighbour_distance_index = 0;
					for ( int j = 0; j < neighbour_size_max; ++j ) {
						const float dist = neighbours_distance_squared[j];
						if ( dist > max_neighbour_distance ) {
							max_neighbour_distance = dist;
							max_neighbour_distance_index = j;
						}
					}
					if (max_neighbour_distance>d2) {
						neighbours[max_neighbour_distance_index]          = cur_index;
						neighbours_distance_squared[max_neighbour_distance_index] = d2;
						neighbours_offset[max_neighbour_distance_index]   = offset;
						neighbours_position[max_neighbour_distance_index] = s_positions[i];
					}
				}
			}
			++i;
		}

		__syncthreads();
	}

	/// @todo try loop till neighbours_found and remove if <- requires more registers
	for (int i=0; i<neighbour_size_max; ++i) {
		if (neighbours[i]==-1) {
			break;
		}
		neighbours_forward[i] = forwards[ neighbours[i] ];
	}

	bool do_seperation[neighbour_size_max];
	bool do_alignment[neighbour_size_max];
	bool do_cohesion[neighbour_size_max];

	for (int i=0; i<neighbour_size_max; ++i) {

		if (neighbours[i]==-1) {
			do_seperation[i] = false;
			do_alignment[i]  = false;
			do_cohesion[i]   = false;
			continue;
		}
		
		/// @todo remove this if and put it into the calculation below
		if (neighbours_distance_squared[i] < boid_radius*3.0f) {
			do_seperation[i] = true;
			do_alignment[i]  = true;
			do_cohesion[i]   = true;
			continue;
		}

		const Vec3  unitOffset  = neighbours_offset[i] / sqrtf (neighbours_distance_squared[i]);
		const float forwardness = forward.dot (unitOffset);
		
		do_seperation[i] = (forwardness > separationAngle && neighbours_distance_squared[i] <= separationRadius*separationRadius);
		do_alignment[i]  = (forwardness > alignmentAngle  && neighbours_distance_squared[i] <= alignmentRadius*alignmentRadius);
		do_cohesion[i]   = (forwardness > cohesionAngle   && neighbours_distance_squared[i] <= cohesionRadius*cohesionRadius);
	}

	Vec3 separation = { 0.0f, 0.0f, 0.0f };
	Vec3 alignment  = { 0.0f, 0.0f, 0.0f };
	Vec3 cohesion   = { 0.0f, 0.0f, 0.0f };
	
	int influencing_alignment_neighbour_count = 0;
	int influencing_cohesion_neighbour_count  = 0;
	for (int i=0; i<neighbour_size_max; ++i) {
		
		/// @todo avoid obstacles if needed


		if (do_seperation[i]) {
			if ( 0.0f != neighbours_distance_squared[i] ) {
				separation = separation + (neighbours_offset[i] / neighbours_distance_squared[i]);
			} else {
				separation = separation + neighbours_offset[i];
			}
		}

		if (do_alignment[i]) {
			// accumulate sum of neighbor's heading
			alignment = alignment + neighbours_forward[i];

			// count neighbors
			++influencing_alignment_neighbour_count;
		}

		if (do_cohesion[i]) {
			// accumulate sum of neighbor's positions
			cohesion = cohesion + neighbours_position[i];

			// count neighbors
			++influencing_cohesion_neighbour_count;
		}
	}
	
	alignment = alignment - ( forward  * influencing_alignment_neighbour_count );
	cohesion  = cohesion  - ( position * influencing_cohesion_neighbour_count  );

	// apply weights to components (save in variables for annotation)
	const Vec3 separationW = separation.normalize() * separationWeight;
	const Vec3 alignmentW  = alignment.normalize()  * alignmentWeight;
	const Vec3 cohesionW   = cohesion.normalize()   * cohesionWeight;

	steering_results[my_index] = separationW + alignmentW + cohesionW;
}
#endif

#if 0
__global__ void find_neighbours_simulate (const cupp::deviceT::vector< Vec3  > &positions_,
                                          const cupp::deviceT::vector< Vec3  > &forwards_,
                                                cupp::deviceT::vector< Vec3 >  &steering_results_)
{
	// make local copies of our references
	__shared__ cupp::deviceT::vector< Vec3 > positions;
	positions = positions_;
	__shared__ cupp::deviceT::vector< Vec3 > forwards;
	forwards = forwards_;
	__shared__ cupp::deviceT::vector< Vec3 > steering_results;
	steering_results = steering_results_;

	// constants that are needed below
	const unsigned int my_index        = blockIdx.x*blockDim.x + threadIdx.x;
	
	__shared__ unsigned int number_of_boids;
	number_of_boids = gridDim.x*blockDim.x; // number of boids == number of threads

	int neighbours_found = 0;
	
	const Vec3 position = positions[my_index];
	const Vec3 forward  = forwards[my_index];
	
	int   neighbours[neighbour_size_max];
	float neighbours_distance_squared[neighbour_size_max];
	Vec3  neighbours_position[neighbour_size_max];
	
	for (int i=0; i<neighbour_size_max; ++i) {
		neighbours[i]=-1;
	}

	for (int base=0; base < number_of_boids; base+=threads_per_block) {
		__shared__ Vec3 s_positions[threads_per_block];
		s_positions[threadIdx.x] = positions[base + threadIdx.x];
		__syncthreads();

		int i=0;
		while (i < threads_per_block) {
			const Vec3 offset = position - s_positions[i];
			const float d2 = offset.lengthSquared();
			const int cur_index = base + i;
			
			if (d2 < r2 && cur_index != my_index) {
				if (neighbours_found < neighbour_size_max) {
					neighbours[neighbours_found]          = cur_index;
					neighbours_distance_squared[neighbours_found] = d2;
					neighbours_position[neighbours_found] = s_positions[i];
					++neighbours_found;
				} else {
					float max_neighbour_distance = 0.0f;
					int max_neighbour_distance_index = 0;
					for ( int j = 0; j < neighbour_size_max; ++j ) {
						const float dist = neighbours_distance_squared[j];
						if ( dist > max_neighbour_distance ) {
							max_neighbour_distance = dist;
							max_neighbour_distance_index = j;
						}
					}
					if (max_neighbour_distance>d2) {
						neighbours[max_neighbour_distance_index]          = cur_index;
						neighbours_distance_squared[max_neighbour_distance_index] = d2;
						neighbours_position[max_neighbour_distance_index] = s_positions[i];
					}
				}
			}
			++i;
		}

		__syncthreads();
	}

	

	bool do_seperation[neighbour_size_max];
	bool do_alignment[neighbour_size_max];
	bool do_cohesion[neighbour_size_max];

	//const int shared_base = threadIdx.x*neighbour_size_max;
	
	for (int i=0; i<neighbour_size_max; ++i) {

		/// @todo try loop till neighbours_found and remove if <- requires more registers
		if (neighbours[i]==-1) {
			do_seperation[i] = false;
			do_alignment[i]  = false;
			do_cohesion[i]   = false;
			continue;
		}
		
		if (neighbours_distance_squared[i] < boid_radius*3.0f) {
			do_seperation[i] = true;
			do_alignment[i]  = true;
			do_cohesion[i]   = true;
			continue;
		}

		const Vec3  unitOffset  = (position - neighbours_position[i]) / sqrtf (neighbours_distance_squared[i]);
		const float forwardness = forward.dot (unitOffset);
		
		do_seperation[i] = (forwardness > separationAngle && neighbours_distance_squared[i] <= separationRadius*separationRadius);
		do_alignment[i]  = (forwardness > alignmentAngle  && neighbours_distance_squared[i] <= alignmentRadius*alignmentRadius);
		do_cohesion[i]   = (forwardness > cohesionAngle   && neighbours_distance_squared[i] <= cohesionRadius*cohesionRadius);
	}

	Vec3 separation = { 0.0f, 0.0f, 0.0f };
	Vec3 alignment  = { 0.0f, 0.0f, 0.0f };
	Vec3 cohesion   = { 0.0f, 0.0f, 0.0f };
	
	int influencing_alignment_neighbour_count = 0;
	int influencing_cohesion_neighbour_count  = 0;
	for (int i=0; i<neighbour_size_max; ++i) {
		
		/// @todo avoid obstacles if needed


		if (do_seperation[i]) {
			if ( 0.0f != neighbours_distance_squared[i] ) {
				separation = separation + ((position - neighbours_position[i]) / neighbours_distance_squared[i]);
			} else {
				separation = separation + position - neighbours_position[i];
			}
		}

		if (do_alignment[i]) {
			// accumulate sum of neighbor's heading
			alignment = alignment + forwards[ neighbours[i] ];

			// count neighbors
			++influencing_alignment_neighbour_count;
		}

		if (do_cohesion[i]) {
			// accumulate sum of neighbor's positions
			cohesion = cohesion + neighbours_position[i];

			// count neighbors
			++influencing_cohesion_neighbour_count;
		}
	}
	
	alignment = alignment - ( forward  * influencing_alignment_neighbour_count );
	cohesion  = cohesion  - ( position * influencing_cohesion_neighbour_count  );

	// apply weights to components (save in variables for annotation)
	const Vec3 separationW = separation.normalize() * separationWeight;
	const Vec3 alignmentW  = alignment.normalize()  * alignmentWeight;
	const Vec3 cohesionW   = cohesion.normalize()   * cohesionWeight;

	steering_results[my_index] = separationW + alignmentW + cohesionW;
}
#endif

#if 0
__global__ void find_neighbours_simulate (const cupp::deviceT::vector< Vec3  > &positions_,
                                          const cupp::deviceT::vector< Vec3  > &forwards_,
                                                cupp::deviceT::vector< Vec3 >  &steering_results_)
{
	// make local copies of our references
	__shared__ cupp::deviceT::vector< Vec3 > positions;
	positions = positions_;
	__shared__ cupp::deviceT::vector< Vec3 > forwards;
	forwards = forwards_;
	__shared__ cupp::deviceT::vector< Vec3 > steering_results;
	steering_results = steering_results_;

	// constants that are needed below
	const unsigned int my_index        = blockIdx.x*blockDim.x + threadIdx.x;
	
	__shared__ unsigned int number_of_boids;
	number_of_boids = gridDim.x*blockDim.x; // number of boids == number of threads

	int neighbours_found = 0;
	
	const Vec3 position = positions[my_index];
	const Vec3 forward  = forwards[my_index];
	
	int   neighbours[neighbour_size_max];
	float neighbours_distance_squared[neighbour_size_max];
	
	for (int i=0; i<neighbour_size_max; ++i) {
		neighbours[i]=-1;
	}

	for (int base=0; base < number_of_boids; base+=threads_per_block) {
		__shared__ Vec3 s_positions[threads_per_block];
		s_positions[threadIdx.x] = positions[base + threadIdx.x];
		__syncthreads();

		int i=0;
		while (i < threads_per_block) {
			const Vec3 offset = position - s_positions[i];
			const float d2 = offset.lengthSquared();
			const int cur_index = base + i;
			
			if (d2 < r2 && cur_index != my_index) {
				if (neighbours_found < neighbour_size_max) {
					neighbours[neighbours_found]          = cur_index;
					neighbours_distance_squared[neighbours_found] = d2;
					++neighbours_found;
				} else {
					float max_neighbour_distance = 0.0f;
					int max_neighbour_distance_index = 0;
					for ( int j = 0; j < neighbour_size_max; ++j ) {
						const float dist = neighbours_distance_squared[j];
						if ( dist > max_neighbour_distance ) {
							max_neighbour_distance = dist;
							max_neighbour_distance_index = j;
						}
					}
					if (max_neighbour_distance>d2) {
						neighbours[max_neighbour_distance_index]          = cur_index;
						neighbours_distance_squared[max_neighbour_distance_index] = d2;
					}
				}
			}
			++i;
		}

		__syncthreads();
	}

	

	bool do_seperation[neighbour_size_max];
	bool do_alignment[neighbour_size_max];
	bool do_cohesion[neighbour_size_max];

	for (int i=0; i<neighbour_size_max; ++i) {

		/// @todo try loop till neighbours_found and remove if <- requires more registers
		if (neighbours[i]==-1) {
			do_seperation[i] = false;
			do_alignment[i]  = false;
			do_cohesion[i]   = false;
			continue;
		}

		const float dist = neighbours_distance_squared[i];
		
		if ( dist < boid_radius*3.0f) {
			do_seperation[i] = true;
			do_alignment[i]  = true;
			do_cohesion[i]   = true;
			continue;
		}

		const Vec3  unitOffset  = (position - positions[ neighbours[i]]) / sqrtf (dist);
		const float forwardness = forward.dot (unitOffset);
		
		do_seperation[i] = (forwardness > separationAngle && dist <= separationRadius*separationRadius);
		do_alignment[i]  = (forwardness > alignmentAngle  && dist <= alignmentRadius*alignmentRadius);
		do_cohesion[i]   = (forwardness > cohesionAngle   && dist <= cohesionRadius*cohesionRadius);
	}

	Vec3 separation = { 0.0f, 0.0f, 0.0f };
	Vec3 alignment  = { 0.0f, 0.0f, 0.0f };
	Vec3 cohesion   = { 0.0f, 0.0f, 0.0f };
	
	int influencing_alignment_neighbour_count = 0;
	int influencing_cohesion_neighbour_count  = 0;
	for (int i=0; i<neighbour_size_max; ++i) {
		
		/// @todo avoid obstacles if needed


		if (do_seperation[i]) {
			Vec3 temp = positions[neighbours[i]];
			if ( 0.0f != neighbours_distance_squared[i] ) {
				separation = separation + ((position - temp) / (position - temp).lengthSquared());
			} else {
				separation = separation + position - temp;
			}
		}

		if (do_alignment[i]) {
			// accumulate sum of neighbor's heading
			alignment = alignment + forwards[ neighbours[i] ];

			// count neighbors
			++influencing_alignment_neighbour_count;
		}

		if (do_cohesion[i]) {
			// accumulate sum of neighbor's positions
			cohesion = cohesion + positions[neighbours[i]];

			// count neighbors
			++influencing_cohesion_neighbour_count;
		}
	}
	
	alignment = alignment - ( forward  * influencing_alignment_neighbour_count );
	cohesion  = cohesion  - ( position * influencing_cohesion_neighbour_count  );

	// apply weights to components (save in variables for annotation)
	const Vec3 separationW = separation.normalize() * separationWeight;
	const Vec3 alignmentW  = alignment.normalize()  * alignmentWeight;
	const Vec3 cohesionW   = cohesion.normalize()   * cohesionWeight;

	steering_results[my_index] = separationW + alignmentW + cohesionW;
}
#endif

// much more registers and not faster
#if 0
__global__ void find_neighbours_simulate (const cupp::deviceT::vector< Vec3  > &positions_,
                                          const cupp::deviceT::vector< Vec3  > &forwards_,
                                                cupp::deviceT::vector< Vec3 >  &steering_results_)
{
	// make local copies of our references
	__shared__ cupp::deviceT::vector< Vec3 > positions;
	positions = positions_;
	__shared__ cupp::deviceT::vector< Vec3 > forwards;
	forwards = forwards_;
	__shared__ cupp::deviceT::vector< Vec3 > steering_results;
	steering_results = steering_results_;

	// constants that are needed below
	const unsigned int my_index        = blockIdx.x*blockDim.x + threadIdx.x;
	
	__shared__ unsigned int number_of_boids;
	number_of_boids = gridDim.x*blockDim.x; // number of boids == number of threads

	int neighbours_found = 0;
	
	const Vec3 position = positions[my_index];
	const Vec3 forward  = forwards[my_index];
	
	int   neighbours[neighbour_size_max];
	float neighbours_distance_squared[neighbour_size_max];
	
	for (int i=0; i<neighbour_size_max; ++i) {
		neighbours[i]=-1;
	}

	for (int base=0; base < number_of_boids; base+=threads_per_block) {
		__shared__ Vec3 s_positions[threads_per_block];
		s_positions[threadIdx.x] = positions[base + threadIdx.x];
		__syncthreads();

		int i=0;
		while (i < threads_per_block) {
			const Vec3 offset = position - s_positions[i];
			const float d2 = offset.lengthSquared();
			const int cur_index = base + i;
			
			if (d2 < r2 && cur_index != my_index) {
				if (neighbours_found < neighbour_size_max) {
					neighbours[neighbours_found]          = cur_index;
					neighbours_distance_squared[neighbours_found] = d2;
					++neighbours_found;
				} else {
					float max_neighbour_distance = 0.0f;
					int max_neighbour_distance_index = 0;
					for ( int j = 0; j < neighbour_size_max; ++j ) {
						const float dist = neighbours_distance_squared[j];
						if ( dist > max_neighbour_distance ) {
							max_neighbour_distance = dist;
							max_neighbour_distance_index = j;
						}
					}
					if (max_neighbour_distance>d2) {
						neighbours[max_neighbour_distance_index]          = cur_index;
						neighbours_distance_squared[max_neighbour_distance_index] = d2;
					}
				}
			}
			++i;
		}

		__syncthreads();
	}

	

	Vec3 separation = { 0.0f, 0.0f, 0.0f };
	Vec3 alignment  = { 0.0f, 0.0f, 0.0f };
	Vec3 cohesion   = { 0.0f, 0.0f, 0.0f };
	
	int influencing_alignment_neighbour_count = 0;
	int influencing_cohesion_neighbour_count  = 0;

	for (int i=0; i<neighbours_found; ++i) {
		
		const float dist = neighbours_distance_squared[i];
		const int index = neighbours[i];
		
		const Vec3  unitOffset  = (position - positions[index]) / sqrtf (dist);
		const float forwardness = forward.dot (unitOffset);
		
		bool do_seperation = dist < boid_radius*3.0f || (forwardness > separationAngle && dist <= separationRadius*separationRadius);
		bool do_alignment  = dist < boid_radius*3.0f || (forwardness > alignmentAngle  && dist <= alignmentRadius*alignmentRadius);
		bool do_cohesion   = dist < boid_radius*3.0f || (forwardness > cohesionAngle   && dist <= cohesionRadius*cohesionRadius);
	
		if (do_seperation) {
			Vec3 temp = positions[index];
			if ( 0.0f != dist ) {
				separation = separation + ((position - temp) / dist);
			} else {
				separation = separation + position - temp;
			}
		}

		if (do_alignment) {
			// accumulate sum of neighbor's heading
			alignment = alignment + forwards[ index ];

			// count neighbors
			++influencing_alignment_neighbour_count;
		}

		if (do_cohesion) {
			// accumulate sum of neighbor's positions
			cohesion = cohesion + positions[index];

			// count neighbors
			++influencing_cohesion_neighbour_count;
		}
	}

	alignment = alignment - ( forward  * influencing_alignment_neighbour_count );
	cohesion  = cohesion  - ( position * influencing_cohesion_neighbour_count  );

	// apply weights to components (save in variables for annotation)
	const Vec3 separationW = separation.normalize() * separationWeight;
	const Vec3 alignmentW  = alignment.normalize()  * alignmentWeight;
	const Vec3 cohesionW   = cohesion.normalize()   * cohesionWeight;

	steering_results[my_index] = separationW + alignmentW + cohesionW;
}
#endif



// Paper version - recalculate
__global__ void find_neighbours_simulate (const cupp::deviceT::vector< Vec3  > &positions_,
                                          const cupp::deviceT::vector< Vec3  > &forwards_,
                                                cupp::deviceT::vector< Vec3 >  &steering_results_)
{
	// make local copies of our references
	__shared__ cupp::deviceT::vector< Vec3 > positions;
	positions = positions_;
	__shared__ cupp::deviceT::vector< Vec3 > forwards;
	forwards = forwards_;
	__shared__ cupp::deviceT::vector< Vec3 > steering_results;
	steering_results = steering_results_;

	// constants that are needed below
	const unsigned int my_index        = blockIdx.x*blockDim.x + threadIdx.x;
	
	__shared__ unsigned int number_of_boids;
	number_of_boids = gridDim.x*blockDim.x; // number of boids == number of threads

	int neighbours_found = 0;
	
	const Vec3 position = positions[my_index];
	const Vec3 forward  = forwards[my_index];
	
	int   neighbours[neighbour_size_max];
	float neighbours_distance_squared[neighbour_size_max];
	
	for (int i=0; i<neighbour_size_max; ++i) {
		neighbours[i]=-1;
	}

	
	for (int base=0; base < number_of_boids; base+=threads_per_block) {
		__shared__ Vec3 s_positions[threads_per_block];
		s_positions[threadIdx.x] = positions[base + threadIdx.x];
		__syncthreads();

		int i=0;
		#pragma unroll 64
		while (i < threads_per_block) {
			const Vec3 offset = position - s_positions[i];
			const float d2 = offset.lengthSquared();
			const int cur_index = base + i;
			
			if (d2 < r2 && cur_index != my_index) {
				if (neighbours_found < neighbour_size_max) {
					neighbours[neighbours_found]          = cur_index;
					neighbours_distance_squared[neighbours_found] = d2;
					++neighbours_found;
				} else {
					float max_neighbour_distance = 0.0f;
					int max_neighbour_distance_index = 0;
					for ( int j = 0; j < neighbour_size_max; ++j ) {
						const float dist = neighbours_distance_squared[j];
						if ( dist > max_neighbour_distance ) {
							max_neighbour_distance = dist;
							max_neighbour_distance_index = j;
						}
					}
					if (max_neighbour_distance>d2) {
						neighbours[max_neighbour_distance_index]          = cur_index;
						neighbours_distance_squared[max_neighbour_distance_index] = d2;
					}
				}
			}
			++i;
		}

		__syncthreads();
	}

	

	bool do_seperation[neighbour_size_max];
	bool do_alignment[neighbour_size_max];
	bool do_cohesion[neighbour_size_max];

	for (int i=0; i<neighbour_size_max; ++i) {

		/// @todo try loop till neighbours_found and remove if <- requires more registers
		if (neighbours[i]==-1) {
			do_seperation[i] = false;
			do_alignment[i]  = false;
			do_cohesion[i]   = false;
			continue;
		}

		const float dist = neighbours_distance_squared[i];
		
		if ( dist < boid_radius*3.0f ) {
			do_seperation[i] = true;
			do_alignment[i]  = true;
			do_cohesion[i]   = true;
			continue;
		}

		const Vec3  unitOffset  = (position - positions[ neighbours[i]]) / sqrtf (dist);
		const float forwardness = forward.dot (unitOffset);
		
		do_seperation[i] = (forwardness > separationAngle && dist <= separationRadius*separationRadius);
		do_alignment[i]  = (forwardness > alignmentAngle  && dist <= alignmentRadius*alignmentRadius);
		do_cohesion[i]   = (forwardness > cohesionAngle   && dist <= cohesionRadius*cohesionRadius);
	}

	Vec3 separation = { 0.0f, 0.0f, 0.0f };
	Vec3 alignment  = { 0.0f, 0.0f, 0.0f };
	Vec3 cohesion   = { 0.0f, 0.0f, 0.0f };
	
	int influencing_alignment_neighbour_count = 0;
	int influencing_cohesion_neighbour_count  = 0;
	for (int i=0; i<neighbour_size_max; ++i) {
		
		/// @todo avoid obstacles if needed

		int index = neighbours[i];
		if (do_seperation[i]) {
			Vec3 temp = position - positions[index];
			if ( 0.0f != temp.lengthSquared() ) {
				separation = separation + (temp / temp.lengthSquared());
			} else {
				separation = separation + temp;
			}
		}

		if (do_alignment[i]) {
			// accumulate sum of neighbor's heading
			alignment = alignment + forwards[ index ];

			// count neighbors
			++influencing_alignment_neighbour_count;
		}

		if (do_cohesion[i]) {
			// accumulate sum of neighbor's positions
			cohesion = cohesion + positions[index];

			// count neighbors
			++influencing_cohesion_neighbour_count;
		}
	}
	
	alignment = alignment - ( forward  * influencing_alignment_neighbour_count );
	cohesion  = cohesion  - ( position * influencing_cohesion_neighbour_count  );

	// apply weights to components (save in variables for annotation)
	const Vec3 separationW = separation.normalize() * separationWeight;
	const Vec3 alignmentW  = alignment.normalize()  * alignmentWeight;
	const Vec3 cohesionW   = cohesion.normalize()   * cohesionWeight;

	steering_results[my_index] = separationW + alignmentW + cohesionW;
}


simulate_kernelT get_find_neighbours_simulate_kernel() {
	return (simulate_kernelT)find_neighbours_simulate;
}
