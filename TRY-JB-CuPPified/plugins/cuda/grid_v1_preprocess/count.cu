#include "hip/hip_runtime.h"
#include "cupp/common.h"
#include "cupp/deviceT/vector.h"

#include "OpenSteer/CuPPConfig.h"
#include "OpenSteer/kernels.h"

#include "OpenSteer/deviceT/Vec3.h"
#include "ds/deviceT/gpu_grid.h"

using OpenSteer::deviceT::Vec3;

__global__ void v1_count(
                           const cupp::deviceT::vector<Vec3>  & positions,
                                 ds::deviceT::gpu_grid        & grid,
                                 unsigned int                   nb_of_boids
                        ) {
#if 0
	const unsigned int my_index        = blockIdx.x*blockDim.x + threadIdx.x;
	if (my_index >= nb_of_boids) return;
	
	// cell index finden
	const Vec3 pos = positions[my_index];
	const int cell = grid.get_index(grid.get_cell_index(pos));
	
	// atomic + 1
	atomicAdd(grid.index_ + cell, 1);
#endif

	const int my_index  = blockIdx.x*threads_per_block + threadIdx.x;
	const int my_cell_x = my_index % grid_size;
	const int my_cell_y = (my_index / grid_size) % grid_size;
	const int my_cell_z = my_index / grid_size / grid_size;

	const float low_x  = my_cell_x * grid.cell_size_;
	const float high_x = (my_cell_x+1) * grid.cell_size_;
	
	const float low_y  = my_cell_y * grid.cell_size_;
	const float high_y = (my_cell_y+1) * grid.cell_size_;
	
	const float low_z  = my_cell_z * grid.cell_size_;
	const float high_z = (my_cell_z+1) * grid.cell_size_;
	
	int count = 0;
	int sh_base = 0;
	
	__shared__ Vec3 sh_positions[threads_per_block];

	const float world_size = grid.world_size_;


	while (sh_base < nb_of_boids) {
		sh_positions[threadIdx.x] = positions[sh_base + threadIdx.x] + world_size;
		__syncthreads();
		
		for (int i=0; i<threads_per_block; ++i) {
			Vec3 &p = sh_positions[i];
			if (p.x() >= low_x && p.x() < high_x &&
			    p.y() >= low_y && p.y() < high_y &&
			    p.z() >= low_z && p.z() < high_z) {
				++count;
			}
		}

		sh_base += threads_per_block;

		__syncthreads();
	}
	
	grid.index_[my_index]=count;

}

v1_countT get_v1_count_kernel() {
	return (v1_countT)v1_count;
}
